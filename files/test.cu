
#include <hip/hip_runtime.h>
#include <iostream>

__global__ void vectorAdd(int *a, int *b, int *c, int n) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < n) {
        c[i] = a[i] + b[i];
    }
}

int GPUTejas_main(int argc, char **argv) { // Renamed main function
    int n = 1024;
    int *a, *b, *c;
    int *d_a, *d_b, *d_c;

    // Memory allocation and initialization code...
    a = new int[n];
    b = new int[n];
    c = new int[n];

    for (int i = 0; i < n; ++i) {
        a[i] = i;
        b[i] = 2 * i;
    }

    hipMalloc((void**)&d_a, n * sizeof(int));
    hipMalloc((void**)&d_b, n * sizeof(int));
    hipMalloc((void**)&d_c, n * sizeof(int));

    hipMemcpy(d_a, a, n * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_b, b, n * sizeof(int), hipMemcpyHostToDevice);

    vectorAdd<<<(n + 255) / 256, 256>>>(d_a, d_b, d_c, n);

    hipMemcpy(c, d_c, n * sizeof(int), hipMemcpyDeviceToHost);

    // Cleanup and deallocation code...
    delete[] a;
    delete[] b;
    delete[] c;
    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);

    return 0;
}
